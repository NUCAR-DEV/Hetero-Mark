#include "hip/hip_runtime.h"
/* Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:Northeastern University Computer Architecture Research (NUCAR)
 * Group, Northeastern University, http://www.ece.neu.edu/groups/nucar/
 *
 * Author: Carter McCardwell (carter@mccardwell.net, cmccardw@ece.neu.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 *  with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense, and/
 * or sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimers. Redistributions in binary
 *   form must reproduce the above copyright notice, this list of conditions and
 *   the following disclaimers in the documentation and/or other materials
 *   provided with the distribution. Neither the names of NUCAR, Northeastern
 *   University, nor the names of its contributors may be used to endorse or
 *   promote products derived from this Software without specific prior written
 *   permission.
 *
 *   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *   IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *   CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *   LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 *   FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 *   DEALINGS WITH THE SOFTWARE.
 */

#include "src/aes/cuda/aes_cuda_benchmark.h"

#include <cstring>
#include <memory>
#include <string>

void AesCudaBenchmark::Initialize() {
  AesBenchmark::Initialize();

  hipMalloc(&d_ciphertext_, text_length_ * sizeof(uint8_t));
  hipMalloc(&d_key_, kExpandedKeyLengthInBytes);
  hipMalloc(&d_s_, 256 * sizeof(uint8_t));
}

__device__ void AddRoundKeyGpu(uint8_t *state, uint32_t *exp_key, int offset) {
  uint8_t *key_bytes = reinterpret_cast<uint8_t *>(exp_key) + 16 * offset;
  state[0] ^= key_bytes[3];
  state[1] ^= key_bytes[2];
  state[2] ^= key_bytes[1];
  state[3] ^= key_bytes[0];
  state[4] ^= key_bytes[7];
  state[5] ^= key_bytes[6];
  state[6] ^= key_bytes[5];
  state[7] ^= key_bytes[4];
  state[8] ^= key_bytes[11];
  state[9] ^= key_bytes[10];
  state[10] ^= key_bytes[9];
  state[11] ^= key_bytes[8];
  state[12] ^= key_bytes[15];
  state[13] ^= key_bytes[14];
  state[14] ^= key_bytes[13];
  state[15] ^= key_bytes[12];
}

__device__ void SubBytesGpu(uint8_t *state, uint8_t *s) {
  state[0] = s[state[0]];
  state[1] = s[state[1]];
  state[2] = s[state[2]];
  state[3] = s[state[3]];
  state[4] = s[state[4]];
  state[5] = s[state[5]];
  state[6] = s[state[6]];
  state[7] = s[state[7]];
  state[8] = s[state[8]];
  state[9] = s[state[9]];
  state[10] = s[state[10]];
  state[11] = s[state[11]];
  state[12] = s[state[12]];
  state[13] = s[state[13]];
  state[14] = s[state[14]];
  state[15] = s[state[15]];
}

__device__ void ShiftRowsGpu(uint8_t *state) {
  uint8_t new_state[16];
  new_state[0] = state[0];
  new_state[1] = state[5];
  new_state[2] = state[10];
  new_state[3] = state[15];
  new_state[4] = state[4];
  new_state[5] = state[9];
  new_state[6] = state[14];
  new_state[7] = state[3];
  new_state[8] = state[8];
  new_state[9] = state[13];
  new_state[10] = state[2];
  new_state[11] = state[7];
  new_state[12] = state[12];
  new_state[13] = state[1];
  new_state[14] = state[6];
  new_state[15] = state[11];

  state[0] = new_state[0];
  state[1] = new_state[1];
  state[2] = new_state[2];
  state[3] = new_state[3];
  state[4] = new_state[4];
  state[5] = new_state[5];
  state[6] = new_state[6];
  state[7] = new_state[7];
  state[8] = new_state[8];
  state[9] = new_state[9];
  state[10] = new_state[10];
  state[11] = new_state[11];
  state[12] = new_state[12];
  state[13] = new_state[13];
  state[14] = new_state[14];
  state[15] = new_state[15];
}

__device__ void MixColumnsGpu(uint8_t *state) {
  for (int i = 0; i < 4; i++) {
    uint8_t *word = state + 4 * i;
    uint8_t a[4];
    uint8_t b[4];
    uint8_t high_bit;
    for (int i = 0; i < 4; i++) {
      a[i] = word[i];
      high_bit = word[i] & 0x80;
      b[i] = word[i] << 1;
      if (high_bit == 0x80) {
        b[i] ^= 0x1b;
      }
    }
    word[0] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
    word[1] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
    word[2] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
    word[3] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0];
  }
}

__global__ void aes_cuda(uint8_t *input, uint32_t *expanded_key, uint8_t *s) {
  uint8_t state[16];

  uint tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = 0; i < 16; i++) {
    state[i] = input[tid * 16 + i];
  }

  AddRoundKeyGpu(state, expanded_key, 0);

  for (int i = 1; i < 14; i++) {
    SubBytesGpu(state, s);
    ShiftRowsGpu(state);
    MixColumnsGpu(state);
    AddRoundKeyGpu(state, expanded_key, i);
  }

  SubBytesGpu(state, s);
  ShiftRowsGpu(state);
  AddRoundKeyGpu(state, expanded_key, 14);

  for (int i = 0; i < 16; i++) {
    input[tid * 16 + i] = state[i];
  }
}

void AesCudaBenchmark::Run() {
  ExpandKey();

  hipMemcpy(d_ciphertext_, plaintext_, text_length_, hipMemcpyHostToDevice);
  hipMemcpy(d_key_, expanded_key_, kExpandedKeyLengthInBytes,
             hipMemcpyHostToDevice);
  hipMemcpy(d_s_, s, 256 * sizeof(uint8_t), hipMemcpyHostToDevice);

  int num_blocks = text_length_ / 16;

  dim3 grid_size(static_cast<size_t>(num_blocks / 64.00));
  dim3 block_size(64);

  cpu_gpu_logger_->GPUOn();
  aes_cuda<<<grid_size, block_size>>>(d_ciphertext_, d_key_, d_s_);

  hipMemcpy(ciphertext_, d_ciphertext_, text_length_, hipMemcpyDeviceToHost);

  cpu_gpu_logger_->GPUOff();
  cpu_gpu_logger_->Summarize();
}

void AesCudaBenchmark::Cleanup() {
  AesBenchmark::Cleanup();
  hipFree(d_ciphertext_);
  hipFree(d_key_);
}
