#include "hip/hip_runtime.h"
/*
 * Hetero-mark
 *
 * Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:
 *   Northeastern University Computer Architecture Research (NUCAR) Group
 *   Northeastern University
 *   http://www.ece.neu.edu/groups/nucar/
 *
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimers.
 *
 *   Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimers in the
 *   documentation and/or other materials provided with the distribution.
 *
 *   Neither the names of NUCAR, Northeastern University, nor the names of
 *   its contributors may be used to endorse or promote products derived
 *   from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include "src/pr/hip/pr_hip_benchmark.h"

#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <cstring>

void PrHipBenchmark::Initialize() {
  PrBenchmark::Initialize();
  hipMalloc(&device_row_offsets, (num_nodes_ + 1) * sizeof(uint32_t));
  hipMalloc(&device_column_numbers, (num_connections_) * sizeof(uint32_t));
  hipMalloc(&device_values, (num_connections_) * sizeof(float));
  hipMalloc(&device_mtx_1, (num_nodes_) * sizeof(float));
  hipMalloc(&device_mtx_2, (num_nodes_) * sizeof(float));
}

__global__ void pr_hip(hipLaunchParm lp, uint32_t *device_row_offsets,
                       uint32_t *device_column_numbers, float *device_values,
                       float *device_mtx_1, float *device_mtx_2) {
  uint tid = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
  uint32_t initialize = device_row_offsets[tid];
  uint32_t limit = device_row_offsets[tid + 1];
  float new_value = 0;
  for (uint32_t j = initialize; j < limit; j++) {
    uint32_t index = device_column_numbers[j];
    new_value += device_values[j] * device_mtx_1[index];
  }
  device_mtx_2[tid] = new_value;
}

void PrHipBenchmark::Run() {
  uint32_t i;

  hipMemcpy(device_row_offsets, row_offsets_,
            (num_nodes_ + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(device_column_numbers, column_numbers_,
            (num_connections_) * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(device_values, values_, (num_connections_) * sizeof(float),
            hipMemcpyHostToDevice);

  dim3 block_size(64);
  dim3 grid_size(num_nodes_ / 64);

  float *temp_mtx =
      reinterpret_cast<float *>(malloc(num_nodes_ * sizeof(float)));
  for (i = 0; i < num_nodes_; i++) {
    temp_mtx[i] = 1.0 / num_nodes_;
  }
  hipMemcpy(device_mtx_1, temp_mtx, num_nodes_ * sizeof(float),
            hipMemcpyHostToDevice);
  free(temp_mtx);

  cpu_gpu_logger_->GPUOn();
  for (i = 0; i < max_iteration_; i++) {
    if (i % 2 == 0) {
      hipLaunchKernel(HIP_KERNEL_NAME(pr_hip), dim3(grid_size),
                      dim3(block_size), 0, 0, device_row_offsets,
                      device_column_numbers, device_values, device_mtx_1,
                      device_mtx_2);
    } else {
      hipLaunchKernel(HIP_KERNEL_NAME(pr_hip), dim3(grid_size),
                      dim3(block_size), 0, 0, device_row_offsets,
                      device_column_numbers, device_values, device_mtx_2,
                      device_mtx_1);
    }
  }

  if (i % 2 != 0) {
    hipMemcpy(page_rank_, device_mtx_1, num_nodes_ * sizeof(float),
              hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(page_rank_, device_mtx_2, num_nodes_ * sizeof(float),
              hipMemcpyDeviceToHost);
  }
  cpu_gpu_logger_->GPUOff();
  cpu_gpu_logger_->Summarize();
}

void PrHipBenchmark::Cleanup() {
  hipFree(device_row_offsets);
  hipFree(device_column_numbers);
  hipFree(device_values);
  hipFree(device_mtx_1);
  hipFree(device_mtx_2);

  PrBenchmark::Cleanup();
}
