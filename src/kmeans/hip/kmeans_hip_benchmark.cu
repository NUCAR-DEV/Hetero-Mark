#include "hip/hip_runtime.h"
/*
 * Hetero-mark
 *
 * Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:
 *   Northeastern University Computer Architecture Research (NUCAR) Group
 *   Northeastern University
 *   http://www.ece.neu.edu/groups/nucar/
 *
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice,
    this list of conditions and the following disclaimers.
 *
 *   Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimers in the
 *   documentation and/or other materials provided with the distribution.
 *
 *   Neither the names of NUCAR, Northeastern University, nor the names of
 *   its contributors may be used to endorse or promote products derived
 *   from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include <math.h>
#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "src/kmeans/hip/kmeans_hip_benchmark.h"

__global__ void kmeans_swap_hip(hipLaunchParm lp, float *feature,
                                float *feature_swap, int npoints,
                                int nfeatures) {
  uint tid = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
  if (tid >= npoints) return;

  for (int i = 0; i < nfeatures; i++)
    feature_swap[i * npoints + tid] = feature[tid * nfeatures + i];
}

__global__ void kmeans_compute_hip(hipLaunchParm lp, float *feature,
                                   float *clusters, int *membership,
                                   int npoints, int nclusters, int nfeatures,
                                   int offset, int size) {
  int point_id = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
  if (point_id >= npoints) return;

  int index = 0;
  if (point_id < npoints) {
    float min_dist = FLT_MAX;
    for (int i = 0; i < nclusters; i++) {
      float dist = 0;
      float ans = 0;
      for (int l = 0; l < nfeatures; l++) {
        ans += (feature[l * npoints + point_id] - clusters[i * nfeatures + l]) *
               (feature[l * npoints + point_id] - clusters[i * nfeatures + l]);
      }

      dist = ans;
      if (dist < min_dist) {
        min_dist = dist;
        index = i;
      }
    }
    membership[point_id] = index;
  }

  return;
}

void KmeansHipBenchmark::Initialize() {
  KmeansBenchmark::Initialize();

  InitializeBuffers();
}

void KmeansHipBenchmark::InitializeBuffers() {
  hipMalloc(&device_membership_, num_points_ * sizeof(int));
  hipMalloc(&device_features_, num_points_ * num_features_ * sizeof(float));
  hipMalloc(&device_features_swap_,
            num_points_ * num_features_ * sizeof(float));
}

void KmeansHipBenchmark::CreateTemporaryMemory() {
  hipMalloc(&device_clusters_, num_clusters_ * num_features_ * sizeof(float));
}

void KmeansHipBenchmark::FreeTemporaryMemory() { hipFree(device_clusters_); }

void KmeansHipBenchmark::Clustering() {
  min_rmse_ = FLT_MAX;
  membership_ = new int[num_points_];

  // Sweep k from min to max_clusters_ to find the best number of clusters
  for (num_clusters_ = min_num_clusters_; num_clusters_ <= max_num_clusters_;
       num_clusters_++) {
    // Sanity check: cannot have more clusters than points
    if (num_clusters_ > num_points_) break;

    CreateTemporaryMemory();
    TransposeFeatures();
    KmeansClustering(num_clusters_);

    float rmse = CalculateRMSE();
    if (rmse < min_rmse_) {
      min_rmse_ = rmse;
      best_num_clusters_ = num_clusters_;
    }
    FreeTemporaryMemory();
  }

  delete[] membership_;
}

void KmeansHipBenchmark::TransposeFeatures() {
  hipMemcpy(device_features_, host_features_,
            num_points_ * num_features_ * sizeof(float), hipMemcpyHostToDevice);

  dim3 block_size(64);
  dim3 grid_size((num_points_ + block_size.x - 1) / block_size.x);

  cpu_gpu_logger_->GPUOn();
  hipLaunchKernel(HIP_KERNEL_NAME(kmeans_swap_hip), dim3(grid_size),
                  dim3(block_size), 0, 0, device_features_,
                  device_features_swap_, num_points_, num_features_);
  hipDeviceSynchronize();
  cpu_gpu_logger_->GPUOff();
}

void KmeansHipBenchmark::KmeansClustering(unsigned num_clusters) {
  unsigned num_iteration = 0;

  // that would guarantee a cluster without points
  if (num_clusters > num_points_) {
    printf("# of clusters cannot be less than # of points\n");
    exit(-1);
  }

  InitializeClusters(num_clusters);
  InitializeMembership();

  // iterate until converge
  do {
    UpdateMembership(num_clusters);
    UpdateClusterCentroids(num_clusters);
    num_iteration++;
  } while ((delta_ > 0) && (num_iteration < 500));

  printf("iterated %d times\n", num_iteration);
}

void KmeansHipBenchmark::UpdateMembership(unsigned num_clusters) {
  int *new_membership = new int[num_points_];

  dim3 block_size(64);
  dim3 grid_size((num_points_ + block_size.x - 1) / block_size.x);

  hipMemcpy(device_clusters_, clusters_,
            num_clusters_ * num_features_ * sizeof(float),
            hipMemcpyHostToDevice);

  int size = 0;
  int offset = 0;

  cpu_gpu_logger_->GPUOn();
  hipLaunchKernel(HIP_KERNEL_NAME(kmeans_compute_hip), dim3(grid_size),
                  dim3(block_size), 0, 0, device_features_swap_,
                  device_clusters_, device_membership_, num_points_,
                  num_clusters_, num_features_, offset, size);
  hipDeviceSynchronize();
  cpu_gpu_logger_->GPUOff();

  hipMemcpy(new_membership, device_membership_, num_points_ * sizeof(int),
            hipMemcpyDeviceToHost);

  cpu_gpu_logger_->CPUOn();
  delta_ = 0.0f;
  for (unsigned int i = 0; i < num_points_; i++) {
    /* printf("number %d, merbership %d\n", i, new_membership[i]); */
    if (new_membership[i] != membership_[i]) {
      delta_++;
      membership_[i] = new_membership[i];
    }
  }
  cpu_gpu_logger_->CPUOff();
}

void KmeansHipBenchmark::Run() { 
  Clustering(); 
  cpu_gpu_logger_->Summarize();
}

void KmeansHipBenchmark::Cleanup() {
  hipFree(device_features_);
  hipFree(device_features_swap_);
  hipFree(device_membership_);
}
