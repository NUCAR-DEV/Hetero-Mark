#include "hip/hip_runtime.h"
/*
 * Hetero-mark
 *
 * Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:
 *   Northeastern University Computer Architecture Research (NUCAR) Group
 *   Northeastern University
 *   http://www.ece.neu.edu/groups/nucar/
 *
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice,
    this list of conditions and the following disclaimers.
 *
 *   Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimers in the
 *   documentation and/or other materials provided with the distribution.
 *
 *   Neither the names of NUCAR, Northeastern University, nor the names of
 *   its contributors may be used to endorse or promote products derived
 *   from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include <math.h>
#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include "src/kmeans/cuda/kmeans_cuda_benchmark.h"

__global__ void kmeans_swap_cuda(float *feature, float *feature_swap,
                                 int npoints, int nfeatures) {
  uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= npoints) return;

  for (int i = 0; i < nfeatures; i++)
    feature_swap[i * npoints + tid] = feature[tid * nfeatures + i];
}

__global__ void kmeans_compute_cuda(float *feature, float *clusters,
                                    int *membership, int npoints, int nclusters,
                                    int nfeatures, int offset, int size) {
  uint32_t point_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_id > npoints) return;

  int index = 0;
  if (point_id < npoints) {
    float min_dist = FLT_MAX;
    for (int i = 0; i < nclusters; i++) {
      float dist = 0;
      float ans = 0;
      for (int l = 0; l < nfeatures; l++) {
        ans += (feature[l * npoints + point_id] - clusters[i * nfeatures + l]) *
               (feature[l * npoints + point_id] - clusters[i * nfeatures + l]);
      }

      dist = ans;
      if (dist < min_dist) {
        min_dist = dist;
        index = i;
      }
    }
    membership[point_id] = index;
  }

  return;
}

void KmeansCudaBenchmark::Initialize() {
  KmeansBenchmark::Initialize();

  InitializeBuffers();
}

void KmeansCudaBenchmark::InitializeBuffers() {
  hipMalloc(&device_membership_, num_points_ * sizeof(int));
  hipMalloc(&device_features_, num_points_ * num_features_ * sizeof(float));
  hipMalloc(&device_features_swap_,
             num_points_ * num_features_ * sizeof(float));
}

void KmeansCudaBenchmark::CreateTemporaryMemory() {
  hipMalloc(&device_clusters_, num_clusters_ * num_features_ * sizeof(float));
}

void KmeansCudaBenchmark::FreeTemporaryMemory() { hipFree(device_clusters_); }

void KmeansCudaBenchmark::Clustering() {
  min_rmse_ = FLT_MAX;
  membership_ = new int[num_points_];

  // Sweep k from min to max_clusters_ to find the best number of clusters
  for (num_clusters_ = min_num_clusters_; num_clusters_ <= max_num_clusters_;
       num_clusters_++) {
    // Sanity check: cannot have more clusters than points
    if (num_clusters_ > num_points_) break;

    CreateTemporaryMemory();
    TransposeFeatures();
    KmeansClustering(num_clusters_);

    float rmse = CalculateRMSE();
    if (rmse < min_rmse_) {
      min_rmse_ = rmse;
      best_num_clusters_ = num_clusters_;
    }
    FreeTemporaryMemory();
  }

  delete[] membership_;
}

void KmeansCudaBenchmark::TransposeFeatures() {
  hipMemcpy(device_features_, host_features_,
             num_points_ * num_features_ * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 block_size(64);
  dim3 grid_size((num_points_ + block_size.x - 1) / block_size.x);

  cpu_gpu_logger_->GPUOn();
  kmeans_swap_cuda<<<grid_size, block_size>>>(
      device_features_, device_features_swap_, num_points_, num_features_);
  cpu_gpu_logger_->GPUOff();
}

void KmeansCudaBenchmark::KmeansClustering(unsigned num_clusters) {
  unsigned num_iteration = 0;

  // that would guarantee a cluster without points
  if (num_clusters > num_points_) {
    printf("# of clusters cannot be less than # of points\n");
    exit(-1);
  }

  InitializeClusters(num_clusters);
  InitializeMembership();

  // iterate until converge
  do {
    UpdateMembership(num_clusters);
    UpdateClusterCentroids(num_clusters);
    num_iteration++;
  } while ((delta_ > 0) && (num_iteration < 500));

  printf("iterated %d times\n", num_iteration);
}

void KmeansCudaBenchmark::UpdateMembership(unsigned num_clusters) {
  int *new_membership = new int[num_points_];

  dim3 block_size(64);
  dim3 grid_size((num_points_ + block_size.x - 1) / block_size.x);

  hipMemcpy(device_clusters_, clusters_,
             num_clusters_ * num_features_ * sizeof(float),
             hipMemcpyHostToDevice);

  int size = 0;
  int offset = 0;

  cpu_gpu_logger_->GPUOn();
  kmeans_compute_cuda<<<grid_size, block_size>>>(
      device_features_swap_, device_clusters_, device_membership_, num_points_,
      num_clusters_, num_features_, offset, size);
  cpu_gpu_logger_->GPUOff();

  hipMemcpy(new_membership, device_membership_, num_points_ * sizeof(int),
             hipMemcpyDeviceToHost);

  cpu_gpu_logger_->CPUOn();
  delta_ = 0.0f;
  for (unsigned int i = 0; i < num_points_; i++) {
    /* printf("number %d, merbership %d\n", i, new_membership[i]); */
    if (new_membership[i] != membership_[i]) {
      delta_++;
      membership_[i] = new_membership[i];
    }
  }
  cpu_gpu_logger_->CPUOff();
}

void KmeansCudaBenchmark::Run() { 
  Clustering(); 
  cpu_gpu_logger_->Summarize();
}

void KmeansCudaBenchmark::Cleanup() {
  hipFree(device_membership_);
  hipFree(device_features_);
  hipFree(device_features_swap_);
}
