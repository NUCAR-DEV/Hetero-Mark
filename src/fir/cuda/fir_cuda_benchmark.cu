#include "hip/hip_runtime.h"
/*
 * Hetero-mark
 *
 * Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:
 *   Northeastern University Computer Architecture Research (NUCAR) Group
 *   Northeastern University
 *   http://www.ece.neu.edu/groups/nucar/
 *
 * Author: Yifan Sun (yifansun@coe.neu.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimers.
 *
 *   Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimers in the
 *   documentation and/or other materials provided with the distribution.
 *
 *   Neither the names of NUCAR, Northeastern University, nor the names of
 *   its contributors may be used to endorse or promote products derived
 *   from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBU TORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include "src/fir/cuda/fir_cuda_benchmark.h"

#include <cstdio>
#include <cstdlib>
#include <cstring>

__global__ void fir_cuda(float *input, float *output, float *coeff,
                         float *history, uint32_t num_tap, uint32_t num_data) {
  uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > num_data) return;

  float sum = 0;
  uint32_t i = 0;
  for (i = 0; i < num_tap; i++) {
    if (tid >= i) {
      sum = sum + coeff[i] * input[tid - i];
    } else {
      sum = sum + coeff[i] * history[num_tap - (i - tid)];
    }
  }
  output[tid] = sum;
}

void FirCudaBenchmark::Initialize() {
  FirBenchmark::Initialize();
  InitializeBuffers();
  InitializeData();
}

void FirCudaBenchmark::InitializeBuffers() {
  history_ = reinterpret_cast<float *>(malloc(num_tap_ * sizeof(float)));
  hipMalloc(&input_buffer_, sizeof(float) * num_data_per_block_);
  hipMalloc(&output_buffer_, sizeof(float) * num_data_per_block_);
  hipMalloc(&coeff_buffer_, sizeof(float) * num_tap_);
  hipMalloc(&history_buffer_, sizeof(float) * num_tap_);
}

void FirCudaBenchmark::InitializeData() {
  for (unsigned i = 0; i < num_tap_; i++) {
    history_[i] = 0.0;
  }

  hipMemcpy(history_buffer_, history_, num_tap_ * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(coeff_buffer_, coeff_, num_tap_ * sizeof(float),
             hipMemcpyHostToDevice);
}

void FirCudaBenchmark::Run() {
  unsigned int count = 0;

  for (unsigned i = 0; i < num_tap_; i++) {
    history_[i] = 0.0;
  }
  hipMemcpy(history_buffer_, history_, num_tap_ * sizeof(float),
             hipMemcpyHostToDevice);


  dim3 grid_size(num_data_per_block_ / 64);
  dim3 block_size(64);

  while (count < num_block_) {
    hipMemcpy(input_buffer_, input_ + (count * num_data_per_block_),
               (num_data_per_block_) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(history_buffer_, history_, num_tap_ * sizeof(float),
               hipMemcpyHostToDevice);
    fir_cuda<<<grid_size, block_size>>>(input_buffer_, output_buffer_,
                                        coeff_buffer_, history_buffer_,
                                        num_tap_, num_data_per_block_);
    hipMemcpy(output_ + count * num_data_per_block_, output_buffer_,
               num_data_per_block_ * sizeof(float), hipMemcpyDeviceToHost);

    for (uint32_t i = 0; i < num_tap_; i++) {
      history_[i] = input_[count * num_data_per_block_ + num_data_per_block_ -
                           num_tap_ + i];
    }

    count++;
  }
}

void FirCudaBenchmark::Cleanup() {
  FirBenchmark::Cleanup();
  free(history_);
  hipFree(output_buffer_);
  hipFree(coeff_buffer_);
  hipFree(input_buffer_);
  hipFree(history_buffer_);
}
