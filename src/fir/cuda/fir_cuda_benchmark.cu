#include "hip/hip_runtime.h"
/*
 * Hetero-mark
 *
 * Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:
 *   Northeastern University Computer Architecture Research (NUCAR) Group
 *   Northeastern University
 *   http://www.ece.neu.edu/groups/nucar/
 *
 * Author: Yifan Sun (yifansun@coe.neu.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimers.
 *
 *   Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimers in the
 *   documentation and/or other materials provided with the distribution.
 *
 *   Neither the names of NUCAR, Northeastern University, nor the names of
 *   its contributors may be used to endorse or promote products derived
 *   from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBU TORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include "src/fir/cuda/fir_cuda_benchmark.h"

__global__ void fir_cuda(float *input, float *output, float *coeff,
                         float *history, uint num_tap) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint num_data = gridDim.x * blockDim.x;

  float sum = 0;
  uint i = 0;
  for (i = 0; i < num_tap; i++) {
    if (tid >= i) {
      sum = sum + coeff[i] * input[tid - i];
    } else {
      sum = sum + coeff[i] * history[num_tap - (i - tid)];
    }
  }
  output[tid] = sum;

  __syncthreads();

  if (tid >= num_data - num_tap) {
    history[num_tap - (num_data - tid)] = input[tid];
  }
}

void FirCudaBenchmark::Initialize() {
  FirBenchmark::Initialize();
  InitializeBuffers();
  InitializeData();
}

void FirCudaBenchmark::InitializeBuffers() {
  hipMallocManaged((void **)&input_buffer_,
                    sizeof(float) * num_data_per_block_);
  hipMallocManaged((void **)&output_buffer_,
                    sizeof(float) * num_data_per_block_);
  hipMallocManaged((void **)&coeff_buffer_, sizeof(float) * num_tap_);
  hipMallocManaged((void **)&history_buffer_, sizeof(float) * num_tap_);
}

void FirCudaBenchmark::InitializeData() {
  for (unsigned i = 0; i < num_tap_; i++) {
    coeff_buffer_[i] = coeff_[i];
  }
  for (unsigned i = 0; i < num_tap_; i++) {
    history_buffer_[i] = 0.0;
  }
}

void FirCudaBenchmark::Run() {
  unsigned int count = 0;

  dim3 grid_size(num_data_per_block_ / 64);
  dim3 block_size(64);

  while (count < num_block_) {
    hipMemcpy(input_buffer_, input_ + (count * num_data_per_block_),
               (num_data_per_block_) * sizeof(float), hipMemcpyHostToDevice);
    fir_cuda<<<grid_size, block_size>>>(input_buffer_, output_buffer_,
                                        coeff_buffer_, history_buffer_,
                                        num_tap_);
    hipMemcpy(output_ + count * num_data_per_block_, output_buffer_,
               num_data_per_block_ * sizeof(float), hipMemcpyDeviceToHost);
    count++;
  }
}

void FirCudaBenchmark::Cleanup() {
  FirBenchmark::Cleanup();
  hipFree(output_buffer_);
  hipFree(coeff_buffer_);
  hipFree(input_buffer_);
  hipFree(history_buffer_);
}
