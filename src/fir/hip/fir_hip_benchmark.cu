#include "hip/hip_runtime.h"
/*
 * Hetero-mark
 *
 * Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:
 *   Northeastern University Computer Architecture Research (NUCAR) Group
 *   Northeastern University
 *   http://www.ece.neu.edu/groups/nucar/
 *
 * Author: Yifan Sun (yifansun@coe.neu.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimers.
 *
 *   Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimers in the
 *   documentation and/or other materials provided with the distribution.
 *
 *   Neither the names of NUCAR, Northeastern University, nor the names of
 *   its contributors may be used to endorse or promote products derived
 *   from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBU TORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include "hip/hip_runtime.h"
#include "src/fir/hip/fir_hip_benchmark.h"

__global__ void fir_hip(hipLaunchParm lp, float *input, float *output,
                        float *coeff, float *history, uint32_t num_tap) {
  uint32_t tid = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
  uint32_t num_data = hipGridDim_x * hipBlockDim_x;

  float sum = 0;
  uint32_t i = 0;
  for (i = 0; i < num_tap; i++) {
    if (tid >= i) {
      sum = sum + coeff[i] * input[tid - i];
    } else {
      sum = sum + coeff[i] * history[num_tap - (i - tid)];
    }
  }
  output[tid] = sum;
}

void FirHipBenchmark::Initialize() {
  FirBenchmark::Initialize();
  InitializeBuffers();
  InitializeData();
}

void FirHipBenchmark::InitializeBuffers() {
  history_ = reinterpret_cast<float *>(malloc(num_tap_ * sizeof(float)));
  hipMalloc((void **)&input_buffer_, sizeof(float) * num_data_per_block_);
  hipMalloc((void **)&output_buffer_, sizeof(float) * num_data_per_block_);
  hipMalloc((void **)&coeff_buffer_, sizeof(float) * num_tap_);
  hipMalloc((void **)&history_buffer_, sizeof(float) * num_tap_);
}

void FirHipBenchmark::InitializeData() {
  hipMemcpy(coeff_buffer_, coeff_, num_tap_ * sizeof(float),
            hipMemcpyHostToDevice);

  for (unsigned i = 0; i < num_tap_; i++) {
    history_[i] = 0.0;
  }
  hipMemcpy(history_buffer_, history_, num_tap_ * sizeof(float),
            hipMemcpyHostToDevice);
}

void FirHipBenchmark::Run() {
  unsigned int count = 0;

  dim3 grid_size(num_data_per_block_ / 64);
  dim3 block_size(64);

  while (count < num_block_) {
    hipMemcpy(input_buffer_, input_ + (count * num_data_per_block_),
              (num_data_per_block_) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(history_buffer_, history_, num_tap_ * sizeof(float),
              hipMemcpyHostToDevice);
    hipLaunchKernel(HIP_KERNEL_NAME(fir_hip), dim3(grid_size), dim3(block_size),
                    0, 0, input_buffer_, output_buffer_, coeff_buffer_,
                    history_buffer_, num_tap_);
    hipMemcpy(output_ + count * num_data_per_block_, output_buffer_,
              num_data_per_block_ * sizeof(float), hipMemcpyDeviceToHost);

    for (uint32_t i = 0; i < num_tap_; i++) {
      history_[i] = input_[count * num_data_per_block_ + num_data_per_block_ -
                          num_tap_ + i];
    }

    count++;
  }
}

void FirHipBenchmark::Cleanup() {
  FirBenchmark::Cleanup();
  free(history_);
  hipFree(output_buffer_);
  hipFree(coeff_buffer_);
  hipFree(input_buffer_);
  hipFree(history_buffer_);
}
