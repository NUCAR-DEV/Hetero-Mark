#include "hip/hip_runtime.h"
/*
 * Hetero-mark
 *
 * Copyright (c) 2015 Northeastern University
 * All rights reserved.
 *
 * Developed by:
 *   Northeastern University Computer Architecture Research (NUCAR) Group
 *   Northeastern University
 *   http://www.ece.neu.edu/groups/nucar/
 *
 * Author: Yifan Sun (yifansun@coe.neu.edu)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal with the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 *   Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimers.
 *
 *   Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimers in the
 *   documentation and/or other materials provided with the distribution.
 *
 *   Neither the names of NUCAR, Northeastern University, nor the names of
 *   its contributors may be used to endorse or promote products derived
 *   from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBU TORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include "src/fir/hip/fir_hip_benchmark.h"

#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <cstring>

#include "src/common/memory/hsa_svm_memory_manager.h"

__global__ void fir_hip(hipLaunchParm lp, float *input, float *output,
                        float *coeff, float *history, uint32_t num_tap,
                        uint32_t num_data) {
  uint32_t tid = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
  if (tid > num_data) return;

  float sum = 0;
  uint32_t i = 0;
  for (i = 0; i < num_tap; i++) {
    if (tid >= i) {
      sum = sum + coeff[i] * input[tid - i];
    } else {
      sum = sum + coeff[i] * history[num_tap - (i - tid)];
    }
  }
  output[tid] = sum;
}

void FirHipBenchmark::Initialize() {
  FirBenchmark::Initialize();
  InitializeBuffers();
  InitializeData();
  // mem_manager_.reset(new HsaMemoryManager());
}

void FirHipBenchmark::InitializeBuffers() {
  history_ = reinterpret_cast<float *>(malloc(num_tap_ * sizeof(float)));

  if (mem_type_ != "hsa") {
    hipMalloc(&input_buffer_, sizeof(float) * num_data_per_block_);
    hipMalloc(&output_buffer_, sizeof(float) * num_data_per_block_);
    hipMalloc(&coeff_buffer_, sizeof(float) * num_tap_);
    hipMalloc(&history_buffer_, sizeof(float) * num_tap_);
  } 
}

void FirHipBenchmark::InitializeData() {
  if (mem_type_ != "hsa") {
    hipMemcpy(coeff_buffer_, coeff_, num_tap_ * sizeof(float),
            hipMemcpyHostToDevice);

    hipMemcpy(history_buffer_, history_, num_tap_ * sizeof(float),
            hipMemcpyHostToDevice);
  }
}

void FirHipBenchmark::Run() {
  if (mem_type_ == "hsa") {
    RunMemManager();
  } else {
    HipRun();
  }
}

void FirHipBenchmark::RunMemManager() {
  unsigned int count = 0;

  for (unsigned i = 0; i < num_tap_; i++) {
    history_[i] = 0.0;
  }

  auto dmem_coeff = mem_manager_->Shadow(coeff_, num_tap_ * sizeof(float));
  auto dmem_history = mem_manager_->Shadow(history_, num_tap_ * sizeof(float));

  history_buffer_ = static_cast<float *>(dmem_history->GetDevicePtr());
  coeff_buffer_ = static_cast<float *>(dmem_history->GetDevicePtr());

  dim3 grid_size(num_data_per_block_ / 64);
  dim3 block_size(64);

  while (count < num_block_) {
    auto dmem_input = mem_manager_->Shadow(
        input_ + count * num_data_per_block_, 
        num_data_per_block_ * sizeof(float));
    auto dmem_output = mem_manager_->Shadow(
        output_ + count * num_data_per_block_,
        num_data_per_block_ * sizeof(float));

    input_buffer_ = static_cast<float *>(dmem_input->GetDevicePtr());
    output_buffer_ = static_cast<float *>(dmem_input->GetDevicePtr());

    dmem_input->HostToDevice();
    dmem_history->HostToDevice();

    cpu_gpu_logger_->GPUOn();
    hipLaunchKernel(HIP_KERNEL_NAME(fir_hip), dim3(grid_size), dim3(block_size),
                    0, 0, input_buffer_, output_buffer_, coeff_buffer_,
                    history_buffer_, num_tap_, num_data_per_block_);
    hipDeviceSynchronize();
    cpu_gpu_logger_->GPUOff();

    dmem_output->DeviceToHost();

    for (uint32_t i = 0; i < num_tap_; i++) {
      history_[i] = input_[count * num_data_per_block_ + num_data_per_block_ -
                           num_tap_ + i];
    }

    count++;

    dmem_input->Free();
    dmem_output->Free();
  }
  cpu_gpu_logger_->Summarize();
}

void FirHipBenchmark::HipRun() {
  unsigned int count = 0;

  for (unsigned i = 0; i < num_tap_; i++) {
    history_[i] = 0.0;
  }


  dim3 grid_size(num_data_per_block_ / 64);
  dim3 block_size(64);

  while (count < num_block_) {
    hipMemcpy(input_buffer_, input_ + (count * num_data_per_block_),
              (num_data_per_block_) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(history_buffer_, history_, num_tap_ * sizeof(float),
              hipMemcpyHostToDevice);
    cpu_gpu_logger_->GPUOn();
    hipLaunchKernel(HIP_KERNEL_NAME(fir_hip), dim3(grid_size), dim3(block_size),
                    0, 0, input_buffer_, output_buffer_, coeff_buffer_,
                    history_buffer_, num_tap_, num_data_per_block_);
    hipMemcpy(output_ + count * num_data_per_block_, output_buffer_,
              num_data_per_block_ * sizeof(float), hipMemcpyDeviceToHost);
    cpu_gpu_logger_->GPUOff();

    for (uint32_t i = 0; i < num_tap_; i++) {
      history_[i] = input_[count * num_data_per_block_ + num_data_per_block_ -
                           num_tap_ + i];
    }

    count++;
  }
  cpu_gpu_logger_->Summarize();

}

void FirHipBenchmark::Cleanup() {
  FirBenchmark::Cleanup();
  free(history_);
  hipFree(output_buffer_);
  hipFree(coeff_buffer_);
  hipFree(input_buffer_);
  hipFree(history_buffer_);
}
